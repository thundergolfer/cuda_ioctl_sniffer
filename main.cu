#include <hip/hip_runtime.h>
#include <iostream>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

__global__ void addKernel(int *data, int value, int size) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < size) {
    data[index] += value;
  }
}

int main() {
  const int size = 10;
  int *data;

  // Allocate unified memory – accessible from CPU or GPU
  gpuErrchk(hipMallocManaged(&data, size * sizeof(int)));

  // Initialize array on host
  for (int i = 0; i < size; ++i) {
    data[i] = i;
  }

  // Launch kernel to add 1 to each element in the array
  addKernel<<<1, size>>>(data, 1, size);
  gpuErrchk(hipPeekAtLastError());
  
  // Wait for GPU to finish before accessing on host
  gpuErrchk(hipDeviceSynchronize());

  // Print results
  for (int i = 0; i < size; ++i) {
    std::cout << "data[" << i << "] = " << data[i] << std::endl;
  }

  // Free memory
  gpuErrchk(hipFree(data));

  return 0;
}
